#include "hip/hip_runtime.h"
﻿//#pragma once
#include "hip/hip_runtime.h"
#include ""


#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>

#include <stdio.h>
#include <string.h>
#include "malloc.h"
#include <iostream>
#include <stdlib.h>
//#include "head.h"


#define size1 10000000
#define No1 -1
int ex = 1;

int ndos = 0;
int pdos = 0;
int num_n = 0;
int num_r = 0;

char cc1[] = "[ ";
char cc2[] = " ]";
char cc3[] = " : ";
char cc4[] = "\n";
char cc5[] = "-";
char cc6[] = ">";

typedef struct Property Pro;

typedef struct Lab {
	char lab_key[20];
	char lab_value[20];
	Lab* nextlab;
}Lab;

typedef struct Node {
	char name[20];
	char label[20];
	int firstpro;
	int flag=0;
	int lab_num;
	Lab* lab = NULL;
}Node;
struct Property {
	char name[20];
	int  h;
	int  pre1;
	int  next1;
	int  l;
	int  pre2;
	int  next2;
	int  flag=0;
};
typedef struct Que {
	char str[20];
	int tol;
}Que;

Node* headnode = (Node*)malloc(sizeof(Node) * size1);
Pro* headpro = (Pro*)malloc(sizeof(Pro) * size1);

//thrust::host_vector<Node> headnode;
//thrust::host_vector<Pro> headpro;

//thrust::device_vector<Node> D_headnode;
//thrust::device_vector<Pro> D_headpro;

Node* D_headnode;
Pro* D_headpro;

void fileIO(char filename[]);
void createNode(char nodename[], char nodelabel[]);
int checkName(char nodename[]);
char* cypher(char* ch, int serve);
void createPro(char headname[], char lastname[], char proname[]);
int checkName(char nodename[]);
void And(int* A, int* B)
{
	int i = 0;
	while (i != ndos)
	{
		A[i] = A[i] & B[i];
		i++;
	}
}
void Or(int* A, int* B)
{
	int i = 0;
	while (i != ndos)
	{
		A[i] = A[i] | B[i];
		i++;
	}
}
void strc1(char str1[], char str2[]) {
	int len1 = sizeof(str1);
	int len2 = strlen(str2);
	int i = 0;
	while (i != len1)
	{
		str1[i] = '\0';
		i++;
	}
	i = 0;
	while (i != len2)
	{
		str1[i] = str2[i];
		i++;
	}
}

__device__ int Check(char a[], char b[]) {
	int i = 0;
	while (a[i] == b[i])
	{
		if (a[i] == '\0') {
			return 1;
		}
		i++;
	}
	return 0;
}

__device__ int copy(char a[],char b[]) 
{
	int i = 0;
	while (b[i]!='\0')
	{
		a[i] = b[i];
		i++;
	}
}
//
__global__ void D_checkName(int* D_ndos, Node* n_vec, char D_nodename[], int* D_lag)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_ndos)
	{
		if (Check(n_vec[i].name, D_nodename) && (n_vec[i].flag != 1)) {
			*D_lag = i;
		}
	}
}

__global__ void D_searchNode(int* D_ndos, Node* D_headnode, char D_nodename[], char D_nodelabel[], int* D_lag)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_ndos)
	{
		if (Check(D_headnode[i].name, D_nodename) && Check(D_headnode[i].label, D_nodelabel) && (D_headnode[i].flag != 1)) {
			*D_lag = i;
		}
	}
}

__global__ void D_searchPro_A(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag, int* D_lag1)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].h == *D_lag1) && (D_headpro[i].flag != 1)) {
			D_lag[i] = 1;
		}
		else
		{
			D_lag[i] = No1;
		}
	}
}

__global__ void D_searchPro_B(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag, int* D_lag1)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].l == *D_lag1) && (D_headpro[i].flag != 1)) {
			D_lag[i] = 1;
		}
		else
		{
			D_lag[i] = No1;
		}
	}
}

__global__ void D_searchPro_AB(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag, int* D_lag1, int* D_lag2)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].h == *D_lag1) && (D_headpro[i].l == *D_lag2) && (D_headpro[i].flag != 1)) {
			D_lag[i] = 1;
		}
		else
		{
			D_lag[i] = No1;
		}
	}
}

__global__ void D_searchPro(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].flag != 1)) {
			D_lag[i] = 1;
		}
		else
		{
			D_lag[i] = No1;
		}
	}
}

__global__ void D_deleteNode(int* D_ndos, Node* D_headnode,Pro* D_headpro, char D_nodename[], char D_nodelabel[],int* D_num_r)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_ndos)
	{
		if (Check(D_headnode[i].name, D_nodename) && Check(D_headnode[i].label, D_nodelabel) && (D_headnode[i].flag != 1)) {
			D_headnode[i].flag = 1;
			if (D_headnode[i].firstpro != No1) {
				int j = D_headnode[i].firstpro;
				while (1) {
					if (i == D_headpro[j].h)
					{
						D_headpro[j].flag = 1;
						D_num_r--;
						j = D_headpro[j].next1;
						if (j == No1) {
							break;
						}
					}
					else
					{
						D_headpro[j].flag = 1;
						D_num_r--;
						j = D_headpro[j].next2;
						if (j == No1) {
							break;
						}
					}
				}
			}
		}
	}
}

__global__ void D_deletePro_AB(int* D_pdos, Pro* D_headpro, char D_proname[],  int* D_lag1, int* D_lag2, int* D_num_r)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].h == *D_lag1) && (D_headpro[i].l == *D_lag2) && (D_headpro[i].flag != 1)) {
			D_headpro[i].flag = 1;
			D_num_r--;
		}
	}
}

__global__ void D_deletePro_A(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag1, int* D_num_r)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].h == *D_lag1) && (D_headpro[i].flag != 1)) {
			D_headpro[i].flag = 1;
			D_num_r--;
		}
	}
}

__global__ void D_deletePro_B(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag1, int* D_num_r)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].l == *D_lag1) && (D_headpro[i].flag != 1)) {
			D_headpro[i].flag = 1;
			D_num_r--;
		}
	}
}

__global__ void D_deletePro(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_num_r)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].flag != 1)) {
			D_headpro[i].flag = 1;
			D_num_r--;
		}
	}
}

__global__ void D_updateProname_AB(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag1, int* D_lag2,char D_newname[])
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].h == *D_lag1) && (D_headpro[i].l == *D_lag2) && (D_headpro[i].flag != 1)) {
			copy(D_headpro[i].name,D_newname);
		}
	}
}

__global__ void D_updateProname_A(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag1, char D_newname[]) 
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].h == *D_lag1) && (D_headpro[i].flag != 1)) {
			copy(D_headpro[i].name, D_newname);
		}
	}
}

__global__ void D_updateProname_B(int* D_pdos, Pro* D_headpro, char D_proname[], int* D_lag1, char D_newname[]) 
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].l == *D_lag1) && (D_headpro[i].flag != 1)) {
			copy(D_headpro[i].name, D_newname);
		}
	}
}

__global__ void D_updateProname(int* D_pdos, Pro* D_headpro, char D_proname[],  char D_newname[])
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < *D_pdos) {
		if (Check(D_headpro[i].name, D_proname) && (D_headpro[i].flag != 1)) {
			copy(D_headpro[i].name, D_newname);
		}
	}
}
void createproperty(int i, char labelkey[], char labelvalue[])
{
	Lab newlab;
	strc1(newlab.lab_key, labelkey);
	strc1(newlab.lab_value, labelvalue);
	if (headnode[i].lab==NULL)
	{
		headnode[i].lab = &newlab;
	}
	else
	{
		Lab* next = headnode[i].lab;
		if (strcmp(next->lab_key, labelkey) == 0)
		{
			printf("label already exsit!\n");
				return;
		}
		while (next->nextlab!=NULL)
		{
			next = next->nextlab;
			if (strcmp(next->lab_key, labelkey) == 0)
			{
				printf("label already exsit!\n");
				return;
			}
		}
		(*next).nextlab = &newlab;
	}
	//Lab* head = headnode[i].lab;
	/*Lab* next = headnode[i].lab;;
	while (next != NULL)
	{
		if (strcmp(next->lab_key, labelkey) == 0)
		{
			printf("label already exsit!\n");
			return;
		}
		next = next->nextlab;
	}
	next = &newlab;*/
	headnode[i].lab_num++;
}

void deleleproperty(int i,char labelkey[]) 
{
	if (i==No1)
	{
		printf("no that node!\n");
		return;
	}
	
	if (headnode[i].lab_num==0)
	{
		printf("no that property!\n");
	}
	else if (headnode[i].lab_num == 1)
	{
		if (strcmp(headnode[i].lab->lab_key, labelkey) == 0)
		{
			headnode[i].lab = NULL;
			headnode[i].lab_num--;
			return;
		}
		else
		{
			printf("no that property!\n");
		}
	}
	else
	{
		if (strcmp(headnode[i].lab->lab_key, labelkey) == 0)
		{
			headnode[i].lab = headnode[i].lab->nextlab;
			headnode[i].lab_num--;
			return;
		}
		Lab* head = headnode[i].lab;
		Lab* next = head->nextlab;
		while (next != NULL)
		{
			if (strcmp(next->lab_key, labelkey) == 0)
			{
				(*head).nextlab = (*next).nextlab;
				headnode[i].lab_num--;
				return;
			}
		}
	}
	
}

void Data_createPro(char headname[], char lastname[], char proname[])
{
	printf("ndos: %d\n", ndos);
	printf("headname: %s\t lastname: %s ", headname, lastname);
	headpro[pdos].h = No1;
	headpro[pdos].l = No1;
	headpro[pdos].next1 = No1;
	headpro[pdos].next2 = No1;
	headpro[pdos].pre1 = No1;
	headpro[pdos].pre2 = No1;
	headpro[pdos].flag = 0;
	strc1(headpro[pdos].name, proname);
	/*Pro pro;
	pro.l = No1;
	pro.h = No1;
	pro.next1 = No1;
	pro.next2 = No1;
	pro.pre1 = No1;
	pro.pre2 = No1;
	pro.flag = 0;
	strc1(pro.name, proname);
	headpro.push_back(pro);*/
	//D_headpro.push_back(pro);
	int last, head;
	last = checkName(lastname);
	head = checkName(headname);
	printf("check:%d,%d\n", head, last);
	//printf("yyyyyyyyyyyyyyyyyy\n");
	if ((head == No1) && (last == No1)) {
		createNode(headname, headname);
		createNode(lastname, lastname);
		last = (ndos - 1);
		head = (ndos - 2);
		//printf("9999999999999999\n");
	}
	else if (head == No1) {
		createNode(headname, headname);
		head = (ndos - 1);
		//printf("qqqqqqqqqqqqqq\n");
	}
	else if (last == No1) {
		createNode(lastname, lastname);
		last = (ndos - 1);
		//printf("wwwwwwwwwwwww\n");
	}
	//printf("9zzzzzzzzzz9\n");
	headpro[pdos].h = head;
	if (headnode[head].firstpro == No1)     //Éú³ÉÊ®×ÖÁ´±í
	{
		headnode[head].firstpro = pdos;
	}
	else
	{
		int pi = headnode[head].firstpro;
		int lag = 0;
		while (1)
		{

			if (headpro[pi].h == head)
			{
				if (headpro[pi].next1 == No1)
				{
					headpro[pi].next1 = pdos;
					headpro[pdos].pre1 = pi;
					break;
				}
				pi = headpro[pi].next1;
			}
			else
			{
				if (headpro[pi].next2 == No1)
				{
					headpro[pi].next2 = pdos;
					headpro[pdos].pre1 = pi;
					break;
				}
				pi = headpro[pi].next2;
			}
		}
	}
	headpro[pdos].l = last;
	if (headnode[last].firstpro == No1)
	{
		headnode[last].firstpro = pdos;
	}
	else
	{
		int pi = headnode[last].firstpro;
		int lag = 0;
		while (1)
		{
			if (headpro[pi].h == last)
			{
				if (headpro[pi].next1 == No1)
				{
					headpro[pi].next1 = pdos;
					headpro[pdos].pre2 = pi;
					break;
				}
				pi = headpro[pi].next1;
			}
			else
			{
				if (headpro[pi].next2 == No1)
				{
					headpro[pi].next2 = pdos;
					headpro[pdos].pre2 = pi;
					break;
				}
				pi = headpro[pi].next2;
			}
		}
	}
	pdos++;
	num_r++;

}

void Data_createNode(char nodename[], char labelname[]) 
{
	/*Node node;
	node.firstpro = No1;
	node.flag = 0;
	strc1(node.label, labelname);
	strc1(node.name, nodename);
	headnode.push_back(node);*/
	printf("1111111111111111111111!");
	headnode[ndos].firstpro = No1;
	headnode[ndos].flag = 0;
	headnode[ndos].lab = NULL;
	headnode[ndos].lab_num = 0;
	strc1(headnode[ndos].name, nodename);
	strc1(headnode[ndos].label, labelname);
	ndos++;
	printf("ndos: %d", ndos);
	num_n++;
}

void Datainsert(char node_filename[], char pro_filename[])     //读学姐得数据集  名字和标签为同值，没有特征。
{
	FILE* fn;
	ndos = 0;
	num_n = 0;
	int i = 0;
	int j = 0;
	//fopen_s(&fn, node_filename, "r");
	fn=fopen(node_filename, "r");
	while (!feof(fn))
	{
		int head;
		char headname[20];
		fscanf(fn, "%s\n", headname);
		//sprintf_s(headname, "%d", head);
		Data_createNode(headname, headname);

		printf("nodeline: %d\n", i);
		i++;
	}
	fclose(fn);
	printf("insert node success!!!");
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	//D_headnode = headnode;
	//
	
	FILE* fp;
	pdos = 0;
	num_r = 0;
	//std::string filename = "C:\\Users\\YuWenJian\\Desktop\\task+++";
	//fopen_s(&fp, pro_filename, "r");
	fp=fopen(pro_filename, "r");
	while (!feof(fp))
	{
		int head, last, pro;
		char headname[20], lastname[20], proname[20];
		fscanf(fp, "%s", headname);
		fscanf(fp, "%s", lastname);
		fscanf(fp, "%s", proname);
		//sprintf_s(headname, "%d", head);
		//sprintf_s(lastname, "%d", last);
		//sprintf_s(proname, "%d", pro);
		createPro(headname, lastname, proname);

		printf("proine: %d\n", j);
		j++;
	}

	fclose(fp);
	//D_headnode = headnode;
	//D_headpro = headpro;
	printf("insert pro success!!!");
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
}
	

 

void Datastore(char filename1[],char filename2[])
{
	FILE* fn;
	//if ((fn = fopen(filename1, "w")) != NULL)
	//fopen_s(&fn, filename1, "w");
	fn=fopen(filename1, "w");
	int i = 0;
	while (i != ndos)
	{
		if (headnode[i].flag==0)
		{
			//fprintf(fn, "");
			fprintf(fn, "%s\t%s\t%d\t", headnode[i].name,headnode[i].label, headnode[i].lab_num);

		}
		int j = 0;
		Lab* lab=headnode[i].lab;
		while (j < headnode[i].lab_num)
		{
			fprintf(fn, "%s\t%s\t", lab->lab_key,lab->lab_value);
			lab = lab->nextlab;
			j++;
		}
		fprintf(fn, "\n");
		i++;
	}
	fclose(fn);
	ndos = 0;
	num_n = 0;
	FILE* fp;
	//if ((fn = fopen(filename1, "w")) != NULL)
	fp=fopen( filename2, "w");
	i = 0;
	while (i != pdos)
	{
		if (headpro[i].flag == 0)
		{
			fprintf(fn, "%s\t%s\t%s\n", headnode[headpro[i].h],headnode[headpro[i].l], headpro[i].name);
		}
		i++;
	}
	fclose(fp);
	pdos = 0;
	num_r = 0;
}

void Dataload(char filename1[],char filename2[])
{
	FILE* fn;
	ndos = 0;
	num_n = 0;
	int i = 0;
	int j = 0;
	//fopen_s(&fn, filename1, "r");
	fn=fopen( filename1, "r");
	while (!feof(fn))
	{
		char headname[20];
		char labelname[20];
		int tmp=0;

		fscanf(fn, "%s\t", headname);
		//fscanf_s(fn,"%d",)
		//Lab lab;
		fscanf(fn, "%s\t", labelname);
		//sprintf_s(headname, "%d", head);
		fscanf(fn, "%d\t", &tmp);
		Data_createNode(headname, headname);
		headnode[(ndos-1)].lab_num = tmp;
		//printf("tmp:  %d   headnode[--ndos].lab_num:   %d",tmp,  headnode[(ndos-1)].lab_num);
		int n = 0;
		while (n<headnode[(ndos-1)].lab_num)
		{
			char a[20],b[20];
			fscanf(fn, "%s\t", a);
			fscanf(fn, "%s\t", b);
			createproperty((ndos-1),a,b);
			n++;
			
		}
		printf("nodeline: %d\n", i);
		i++;
	}
	fclose(fn);
	//D_headnode = headnode;
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	//
	printf("ndos:   %d",ndos);
	printf("insert node success!!!");
	FILE* fp;
	pdos = 0;
	num_r = 0;
	//std::string filename = "C:\\Users\\YuWenJian\\Desktop\\task+++";
	//fopen_s(&fp, filename2, "r");
	fp=fopen(filename2, "r");
	while (!feof(fp))
	{
		//int head, last, pro;
		char headname[20], lastname[20], proname[20];
		fscanf(fp, "%s\t", headname);
		fscanf(fp, "%s\t", lastname);
		fscanf(fp, "%s\n", proname);
		//sprintf_s(headname, "%d", head);
		//sprintf_s(lastname, "%d", last);
		//sprintf_s(proname, "%d", pro);
		Data_createPro(headname, lastname, proname);

		printf("proine: %d\n", j);
		j++;

	}
	fclose(fp);
	//D_headnode = headnode;
	//D_headpro = headpro;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	printf("insert pro success!!!");
}
void SubgraphIso()
{

}

void  createNode(char nodename[], char nodelabel[])
{
	if (checkName(nodename)!=No1)
	{
		printf("you need change a name\n");
		return;
	}
	headnode[ndos].firstpro = No1;
	headnode[ndos].flag = 0;
	headnode[ndos].lab_num = 0;
	headnode[ndos].lab = NULL;
	strc1(headnode[ndos].name, nodename);
	strc1(headnode[ndos].label, nodelabel);
	ndos++;
	num_n++;
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	//printf("wwwwwwwwwwwwwwww\n");
	//headnode.push_back(node);
	//printf("jjjjjjjjjjjjjjjj\n");
	//D_headnode = headnode;
	//thrust::fill(codes.begin(), codes.end(), 1);
	//printf("qqqqqqqqqqqqqqqqqqq\n");
	//ndos++;
	//num_n++;
	/*Node node;
	node.firstpro = No1;
	node.flag = 0;
	strc1(node.name, nodename);
	strc1(node.label, nodelabel);
	*/
}
void createPro(char headname[], char lastname[], char proname[])
{
	headpro[pdos].h = No1;
	headpro[pdos].l = No1;
	headpro[pdos].next1 = No1;
	headpro[pdos].next2 = No1;
	headpro[pdos].pre1 = No1;
	headpro[pdos].pre2 = No1;
	headpro[pdos].flag = 0;
	strc1(headpro[pdos].name, proname);
		/*
	Pro pro;
	pro.l = No1;
	pro.next1 = No1;
	pro.next2 = No1;
	pro.pre1 = No1;
	pro.pre2 = No1;
	pro.flag = 0;
	strc1(pro.name, proname);
	headpro.push_back(pro);*/
	//D_headpro.push_back(pro);
	int last, head;
	last = checkName(lastname);
	head = checkName(headname);
	printf("check:%d,%d\n", head, last);
	//printf("yyyyyyyyyyyyyyyyyy\n");
	if ((head == No1) && (last == No1)) {
		createNode(headname, headname);
		createNode(lastname, lastname);
		last = (ndos - 1);
		head = (ndos - 2);
		//printf("9999999999999999\n");
	}
	else if (head == No1) {
		createNode(headname, headname);
		head = (ndos - 1);
		//printf("qqqqqqqqqqqqqq\n");
	}
	else if (last == No1) {
		createNode(lastname, lastname);
		last = (ndos - 1);
		//printf("wwwwwwwwwwwww\n");
	}
	//printf("9zzzzzzzzzz9\n");
	headpro[pdos].h = head;
	if (headnode[head].firstpro == No1)     //Éú³ÉÊ®×ÖÁ´±í
	{
		headnode[head].firstpro = pdos;
	}
	else
	{
		int pi = headnode[head].firstpro;
		int lag = 0;
		while (1)
		{

			if (headpro[pi].h == head)
			{
				if (headpro[pi].next1 == No1)
				{
					headpro[pi].next1 = pdos;
					headpro[pdos].pre1 = pi;
					break;
				}
				pi = headpro[pi].next1;
			}
			else
			{
				if (headpro[pi].next2 == No1)
				{
					headpro[pi].next2 = pdos;
					headpro[pdos].pre1 = pi;
					break;
				}
				pi = headpro[pi].next2;
			}
		}
	}
	headpro[pdos].l = last;
	if (headnode[last].firstpro == No1)
	{
		headnode[last].firstpro = pdos;
	}
	else
	{
		int pi = headnode[last].firstpro;
		int lag = 0;
		while (1)
		{
			if (headpro[pi].h == last)
			{
				if (headpro[pi].next1 == No1)
				{
					headpro[pi].next1 = pdos;
					headpro[pdos].pre2 = pi;
					break;
				}
				pi = headpro[pi].next1;
			}
			else
			{
				if (headpro[pi].next2 == No1)
				{
					headpro[pi].next2 = pdos;
					headpro[pdos].pre2 = pi;
					break;
				}
				pi = headpro[pi].next2;
			}
		}
	}

	pdos++;
	num_r++;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
}

int checkName(char nodename[])                //Nodes with the same name are not allowed 
{
	int* lag = (int*)malloc(sizeof(int));
	*lag = -1;
	int* D_lag, * D_ndos;
	char* D_nodename;
	hipMalloc((void**)&D_ndos, sizeof(int));
	hipMalloc((void**)&D_lag, sizeof(int));
	hipMalloc((void**)&D_nodename, sizeof(char) * 20);
	
	hipMemcpy(D_nodename, nodename, sizeof(char) * 20, hipMemcpyHostToDevice);
	
	hipMemcpy(D_lag, lag, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_ndos, &ndos, sizeof(int), hipMemcpyHostToDevice);
	//printf("77777777777777\n");
	int blocksize;
	int gridsize;
	if (ndos < 512)
	{
		blocksize = ndos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (ndos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Node* n_vec = thrust::raw_pointer_cast(D_headnode.data());
	//
	D_checkName << <dimGrid, dimBlock >> > (D_ndos, D_headnode, D_nodename, D_lag);
	//printf("88888888888888888888\n");
	hipMemcpy(lag, D_lag, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(D_lag);
	hipFree(D_ndos);
	hipFree(D_nodename);
	//hipFree(D_headnode);
	//printf("9999999999999999\n");
	return *lag;
}  


int Searchnode(char nodename[], char nodelabel[])    //cuda c 
{
	int* lag = (int*)malloc(sizeof(int));
	//int* lag = nullptr;
	*lag = -1;
	int* D_lag, * D_ndos;
	char* D_nodename;
	char* D_nodelabel;
	hipMalloc((void**)&D_lag, sizeof(int));
	hipMalloc((void**)&D_ndos, sizeof(int));
	hipMalloc((void**)&D_nodename, sizeof(char) * 20);
	hipMalloc((void**)&D_nodelabel, sizeof(char) * 20);
	//hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_nodename, nodename, sizeof(char) * 20, hipMemcpyHostToDevice);
	hipMemcpy(D_nodelabel, nodelabel, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag, lag, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_ndos, &ndos, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (ndos < 516)
	{
		blocksize = ndos;
		gridsize = 1;
	}
	else
	{
		blocksize = 516;
		gridsize = (ndos + 515) / 516;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
	//Node* n_vec = thrust::raw_pointer_cast(D_headnode.data());
	D_searchNode << <dimGrid, dimBlock >> > (D_ndos, D_headnode, D_nodename, D_nodelabel, D_lag);
	hipMemcpy(lag, D_lag, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(D_lag);
	hipFree(D_ndos);
	hipFree(D_nodename);
	hipFree(D_nodelabel);
	//hipFree(D_headnode);

	return *lag;
	//return 1;
}

void SearchPro_AB(int pro[], char proname[], char A[], char B[])            //cuda c
{
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(A);
	int* lag_2 = (int*)malloc(sizeof(int));
	*lag_2 = checkName(B);
	int* D_lag1;
	int* D_lag2;
	int* D_pdos;
	hipMalloc((void**)&D_lag1, sizeof(int));
	hipMalloc((void**)&D_lag2, sizeof(int));
	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	int* D_lag;
	char* D_proname;
	hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_lag2, lag_2, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_searchPro_AB << <dimGrid, dimBlock >> > (D_pdos,D_headpro, D_proname, D_lag, D_lag1, D_lag2);
	hipMemcpy(pro, D_lag, sizeof(int) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_lag);
	hipFree(D_lag1);
	hipFree(D_lag2);
	hipFree(D_pdos);
	hipFree(D_proname);
	//hipFree(D_headpro);
}
void SearchPro_A(int pro[], char proname[], char A[])                          //cuda c
{
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(A);
	int* D_lag1;
	hipMalloc((void**)&D_lag1, sizeof(int));
	int* D_lag;
	char* D_proname;
	hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	int* D_pdos;
	hipMalloc((void**)&D_pdos, sizeof(int));
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_searchPro_A << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag, D_lag1);
	hipMemcpy(pro, D_lag, sizeof(int) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_lag);
	hipFree(D_lag1);
	//hipFree(D_lag2);
	hipFree(D_pdos);
	hipFree(D_proname);
	//hipFree(D_headpro);
}
void SearchPro_B(int pro[], char proname[], char B[])                           //cuda c
{
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(B);
	int* D_lag1;
	hipMalloc((void**)&D_lag1, sizeof(int));
	int* D_lag;
	char* D_proname;
	hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	int* D_pdos;
	hipMalloc((void**)&D_pdos, sizeof(int));
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_searchPro_B << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag, D_lag1);
	hipMemcpy(pro, D_lag, sizeof(int) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_lag);
	hipFree(D_lag1);
	//hipFree(D_lag2);
	hipFree(D_pdos);
	hipFree(D_proname);
	//hipFree(D_headpro);
}
void SearchPro(int pro[], char proname[])               //cuda c
{
	int* D_lag;
	char* D_proname;
	hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	int* D_pdos;
	hipMalloc((void**)&D_pdos, sizeof(int));
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_searchPro << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag);
	hipMemcpy(pro, D_lag, sizeof(int) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_lag);
	//hipFree(D_lag1);
	//hipFree(D_lag2);
	hipFree(D_pdos);
	hipFree(D_proname);
	//hipFree(D_headpro);
}

void Deletnode(char nodename[], char nodelabel[])
{
	int* D_ndos;
	char* D_nodename;
	char* D_nodelabel;
	int* D_num_r;
	hipMalloc((void**)&D_num_r, sizeof(int));
	hipMalloc((void**)&D_ndos, sizeof(int));
	hipMalloc((void**)&D_nodename, sizeof(char) * 20);
	hipMalloc((void**)&D_nodelabel, sizeof(char) * 20);
	//hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_nodename, nodename, sizeof(char) * 20, hipMemcpyHostToDevice);
	hipMemcpy(D_nodelabel, nodelabel, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	hipMemcpy(D_ndos, &ndos, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_num_r, &num_r, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (ndos < 516)
	{
		blocksize = ndos;
		gridsize = 1;
	}
	else
	{
		blocksize = 516;
		gridsize = (ndos + 515) / 516;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
	//Node* n_vec = thrust::raw_pointer_cast(D_headnode.data());
	D_deleteNode << <dimGrid, dimBlock >> > (D_ndos, D_headnode, D_headpro, D_nodename, D_nodelabel,D_num_r);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipMemcpy(headnode, D_headnode, sizeof(Node) * ndos, hipMemcpyDeviceToHost);
	hipMemcpy(&num_r, D_num_r, sizeof(int), hipMemcpyDeviceToHost);
	num_n--;
	hipFree(D_num_r);
	hipFree(D_ndos);
	hipFree(D_nodename);
	hipFree(D_nodelabel);
}
/*void Deletenode(char nodename[], char nodelabel[])
{
	int lag = Searchnode(nodename, nodelabel);
	headnode[lag].flag = 1;
	if (headnode[lag].firstpro != No1) {
		int j = headnode[lag].firstpro;
		while (1) {
			if (lag == headpro[j].h)
			{
				headpro[j].flag = 1;
				num_r--;
				j = headpro[j].next1;
				if (j == No1) {
					break;
				}
			}
			else
			{
				headpro[j].flag = 1;
				num_r--;
				j = headpro[j].next2;
				if (j == No1) {
					break;
				}
			}
		}
	}
	num_n--;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
}
*/
void DeletPro_AB(int pro[], char proname[], char A[], char B[])
{

	/*SearchPro_AB(pro, proname, A, B);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			headpro[i].flag = 1;
		}
		i++;
	}
	num_r--;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	*/
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(A);
	int* lag_2 = (int*)malloc(sizeof(int));
	*lag_2 = checkName(B);
	int* D_lag1;
	int* D_lag2;
	int* D_pdos;
	
	hipMalloc((void**)&D_lag1, sizeof(int));
	hipMalloc((void**)&D_lag2, sizeof(int));
	hipMalloc((void**)&D_pdos, sizeof(int));
	
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_lag2, lag_2, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int* D_num_r;
	hipMalloc((void**)&D_num_r, sizeof(int));
	hipMemcpy(D_num_r, &num_r, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_deletePro_AB << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname,  D_lag1, D_lag2, D_num_r);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipMemcpy(&num_r,D_num_r,sizeof(int), hipMemcpyDeviceToHost);
	hipFree(D_num_r);
	hipFree(D_lag1);
	hipFree(D_lag2);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void DeletPro_A(int pro[], char proname[], char A[])
{
	/*SearchPro_A(pro, proname, A);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			headpro[i].flag = 1;
		}
		i++;
	}
	num_r--;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	*/
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(A);
	int* D_lag1;
	int* D_pdos;
	hipMalloc((void**)&D_lag1, sizeof(int));
	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int* D_num_r;
	hipMalloc((void**)&D_num_r, sizeof(int));
	hipMemcpy(D_num_r, &num_r, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_deletePro_A << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag1, D_num_r);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipMemcpy(&num_r, D_num_r, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(D_num_r);
	hipFree(D_lag1);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void DeletPro_B(int pro[], char proname[], char B[])
{
	/*SearchPro_B(pro, proname, B);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			headpro[i].flag = 1;
		}
		i++;
	}
	num_r--;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	*/
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(B);
	int* D_lag1;
	int* D_pdos;
	hipMalloc((void**)&D_lag1, sizeof(int));
	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int* D_num_r;
	hipMalloc((void**)&D_num_r, sizeof(int));
	hipMemcpy(D_num_r, &num_r, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_deletePro_B << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag1, D_num_r);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipMemcpy(&num_r, D_num_r, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(D_num_r);
	hipFree(D_lag1);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void DeletPro(int pro[], char proname[])
{
	/*SearchPro(pro, proname);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			headpro[i].flag = 1;
		}
		i++;
	}
	num_r--;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	*/
	char* D_proname;
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	int* D_pdos;
	hipMalloc((void**)&D_pdos, sizeof(int));
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	int* D_num_r;
	hipMalloc((void**)&D_num_r, sizeof(int));
	hipMemcpy(D_num_r, &num_r, sizeof(int), hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);
	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_deletePro << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_num_r);
	hipMemcpy(&num_r, D_num_r, sizeof(int), hipMemcpyDeviceToHost);
	hipFree(D_num_r);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void renew()
{
	char name1[] = "tmp_file1";
	char name2[] = "tmp_file2";
	Datastore(name1,name2);
	Dataload(name1,name2);
	remove(name1);
	remove(name2);
}
void updatenodename(char nodename[], char newname[])
{
	int lag = checkName(nodename);
	int lag1= checkName(newname);
	if (lag==No1)
	{
		printf("This Node not exist!");
		return;
	}
	if (lag1!=No1)
	{
		printf("new name has ready exist!");
		return;
	}
	strc1(headnode[lag].name, newname);
	strc1(headnode[lag].label, newname);
	//hipMalloc((void**)&D_headnode, sizeof(Node) * ndos);
	hipMemcpy(D_headnode, headnode, sizeof(Node) * ndos, hipMemcpyHostToDevice);
	//D_headpro = headpro;
}
void updateProname_AB(char proname[], char A[], char B[], char newname[])
{
	char* D_newname;

	/*int* pro = (int*)malloc(sizeof(int) * pdos);
	SearchPro_AB(pro, proname, A, B);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			strc1(headpro[i].name, newname);
		}
		i++;
	}
	//D_headnode = headnode;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	*/
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(A);
	int* lag_2 = (int*)malloc(sizeof(int));
	*lag_2 = checkName(B);
	int* D_lag1;
	int* D_lag2;
	int* D_pdos;
	hipMalloc((void**)&D_lag1, sizeof(int));
	hipMalloc((void**)&D_lag2, sizeof(int));
	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_newname, sizeof(char) * 20);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_lag2, lag_2, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_newname, newname, sizeof(char) * 20, hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_updateProname_AB << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag1, D_lag2,D_newname);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_newname);
	hipFree(D_lag1);
	hipFree(D_lag2);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void updateProname_A(char proname[], char A[], char newname[])
{
	char* D_newname;
	/*int* pro = (int*)malloc(sizeof(int) * pdos);
	SearchPro_A(pro, proname, A);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			strc1(headpro[i].name, newname);
		}
		i++;
	}
	//D_headnode = headnode;
	//D_headpro = headpro;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	*/
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(A);
	int* D_lag1;
	int* D_pdos;
	hipMalloc((void**)&D_lag1, sizeof(int));
	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_newname, sizeof(char) * 20);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_newname, newname, sizeof(char) * 20, hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_updateProname_A << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag1,  D_newname);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);

	hipFree(D_newname);
	hipFree(D_lag1);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void updateProname_B(char proname[], char B[], char newname[])
{
	char* D_newname;
	/*
	int* pro = (int*)malloc(sizeof(int) * pdos);
	SearchPro_B(pro, proname, B);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			strc1(headpro[i].name, newname);
		}
		i++;
	}
	//D_headnode = headnode;
	//D_headpro = headpro;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	*/
	int* lag_1 = (int*)malloc(sizeof(int));
	*lag_1 = checkName(B);
	int* D_lag1;
	int* D_pdos;
	hipMalloc((void**)&D_lag1, sizeof(int));

	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_newname, sizeof(char) * 20);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);
	hipMemcpy(D_lag1, lag_1, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_newname, newname, sizeof(char) * 20, hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_updateProname_B << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_lag1,  D_newname);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_newname);
	hipFree(D_lag1);
	hipFree(D_pdos);
	hipFree(D_proname);
}
void updateProname(char proname[], char newname[])
{
	char* D_newname;
	/*
	int* pro = (int*)malloc(sizeof(int) * pdos);
	SearchPro(pro, proname);
	int i = 0;
	while (i != pdos)
	{
		if (pro[i] == 1)
		{
			strc1(headpro[i].name, newname);
		}
		i++;
	}
	//D_headnode = headnode;
	//D_headpro = headpro;
	hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	*/
	int* D_pdos;
	hipMalloc((void**)&D_pdos, sizeof(int));
	//hipMalloc((void**)&D_headpro, sizeof(Pro) * pdos);
	//int* D_lag;
	char* D_proname;
	//hipMalloc((void**)&D_lag, sizeof(int) * pdos);
	hipMalloc((void**)&D_newname, sizeof(char) * 20);
	hipMalloc((void**)&D_proname, sizeof(char) * 20);
	hipMemcpy(D_proname, proname, sizeof(char) * 20, hipMemcpyHostToDevice);
	//hipMemcpy(D_headpro, headpro, sizeof(Pro) * pdos, hipMemcpyHostToDevice);
	//hipMemcpy(D_lag, pro, sizeof(int) * pdos, hipMemcpyHostToDevice);

	hipMemcpy(D_pdos, &pdos, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(D_newname, newname, sizeof(char) * 20, hipMemcpyHostToDevice);
	int blocksize;
	int gridsize;
	if (pdos < 512)
	{
		blocksize = pdos;
		gridsize = 1;
	}
	else
	{
		blocksize = 128;
		gridsize = (pdos + 127) / 128;
	}
	dim3 dimBlock(blocksize);
	dim3 dimGrid(gridsize);

	//Pro* p_vec = thrust::raw_pointer_cast(D_headpro.data());
	D_updateProname << <dimGrid, dimBlock >> > (D_pdos, D_headpro, D_proname, D_newname);
	hipMemcpy(headpro, D_headpro, sizeof(Pro) * pdos, hipMemcpyDeviceToHost);
	hipFree(D_newname);
	hipFree(D_pdos);
	hipFree(D_proname);
}

void K_neiber()
{

}

void cypher(char* ch)
{
	int lag = 1;
	while (lag) {
		Que* Token = (Que*)malloc(100 * sizeof(Que));
		int high = 0;
		int i = 0, len = 0;
		while (i < strlen(ch))
		{
			int l = 0;
			while (ch[i] == ' ')
			{
				i++;
			}
			int p = 0;
			while (p < sizeof(Token[high].str))
			{
				Token[high].str[p] = '\0';
				p++;
			}
			while ((ch[i] != ' ') && (ch[i] != '\0'))
			{
				if (ch[i] == ':')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 1;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 1;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == ',')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 2;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 2;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == '[')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 3;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 3;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == ']')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 4;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 4;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == '(')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 5;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 5;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == ')')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 6;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 6;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == '-')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 7;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 7;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == '>')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 8;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 8;
						l = 0;
						i++;
						break;
					}
				}
				if (ch[i] == '=')
				{
					if (l == 1) {
						Token[high].tol = 0;
						high++;
						Token[high].str[0] = ch[i];
						Token[high].tol = 9;
						l = 0;
						i++;
						break;
					}
					else
					{
						Token[high].str[0] = ch[i];
						Token[high].tol = 9;
						l = 0;
						i++;
						break;
					}
				}
				Token[high].str[len] = ch[i];
				i++;
				len++;
				l = 1;
			}
			if (l == 1)
			{
				Token[high].tol = 0;
			}
			high++;
			len = 0;
		}
		char c1[] = "createnode";
		char c2[] = "createrel";
		char c3[] = "matchnode";
		char c4[] = "matchrel";
		char c5[] = "deletenode";
		char c6[] = "deleterel";
		char c7[] = "where";
		char c8[] = "and";
		char c9[] = "or";
		char c10[] = "change";
		char c11[] = "updatenode";
		char c12[] = "updaterel";
		char c13[] = "name";
		char c14[] = "labelname";
		char c15[] = "datainsert";
		char c17[] = "Test";
		char c18[] = "addproperty";
		char c19[] = "clear";
		char c20[] = "Exit";
		char c21[] = "deleteproperty";
		char c22[] = "renew";
		int z = 0;
		if (strcmp(Token[z].str, c1) == 0)    //createnode
		{
			char err[] = "[ input err ]";
			char nodename[20], labelname[20];
			z++;
			if (Token[z].tol == 5)           //(
			{
				z++;
				if (Token[z].tol == 0)    //str
				{
					strc1(nodename, Token[z].str);
					z++;
					if (Token[z].tol == 2)     //,
					{
						z++;
						if (Token[z].tol == 0)     //str
						{
							strc1(labelname, Token[z].str);
							z++;
							if (Token[z].tol == 6) {
								z++;
								if (z == high)
								{
									createNode(nodename, labelname);
									printf("create success!\n");
									printf("node:%d  relation:%d\n", num_n, num_r);
									break;
								}
							}
						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c2) == 0) //createrel
		{
			char err[] = "[ input err! ]";
			char proname[20], A[20], B[20];
			z++;
			if (Token[z].tol == 5)     //(
			{
				z++;
				if (Token[z].tol == 0)     //str
				{
					strc1(A, Token[z].str);
					z++;
					if (Token[z].tol == 7)     //-
					{
						z++;
						if (Token[z].tol == 3)     //[
						{
							z++;
							if (Token[z].tol == 0)      //str
							{
								strc1(proname, Token[z].str);
								z++;
								if (Token[z].tol == 4)      //]
								{
									z++;
									if (Token[z].tol == 7)      //-
									{
										z++;
										if (Token[z].tol == 8)      //>
										{
											z++;
											if (Token[z].tol == 0)      //str
											{
												strc1(B, Token[z].str);
												z++;
												if (Token[z].tol == 6)      //)
												{
													z++;
													if (z == high) {
														//int lag=Create_pro(A, B, proname);
														createPro(A, B, proname);
														printf("create relation success!\n");
														printf("node:%d  relation:%d\n", num_n, num_r);
														break;
													}

												}

											}

										}

									}

								}

							}

						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c3) == 0)//matchnode
		{
			char err[] = "[ input err! ]";
			z++;
			if (Token[z].tol == 0)      //str
			{
				char nodename[20];
				int  inode;
				strc1(nodename, Token[z].str);
				z++;
				if (Token[z].tol == 5)      //(
				{
					z++;
					if (strcmp(Token[z].str, c7) == 0)   //where
					{
						z++;
						if (strcmp(Token[z].str, c14) == 0)     //labelname
						{
							z++;
							if (Token[z].tol == 9)      //=
							{
								z++;
								if (Token[z].tol == 0)      //str
								{
									char labname1[20];
									strc1(labname1, Token[z].str);
									z++;
									if (Token[z].tol == 6)     //)
									{
										z++;
										if (z == high)
										{
											inode = Searchnode(nodename, labname1);
											printf("node:%d  relation:%d\n", num_n, num_r);
											printf("nodname:%s nodelabel:%s ");
											break;
										}

									}

								}

							}

						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c4) == 0)  //match rel
		{
			int* irel = (int*)malloc(sizeof(int) * pdos);
			z++;
			if (Token[z].tol == 5)      //(
			{
				z++;
				if (Token[z].tol == 0)       //str
				{
					char A[20];
					strc1(A, Token[z].str);
					z++;
					if (Token[z].tol == 7)       //-
					{

						z++;
						if (Token[z].tol == 3)      //[
						{
							z++;
							if (Token[z].tol == 0)      //str
							{
								char proname[20];
								strc1(proname, Token[z].str);
								z++;
								if (Token[z].tol == 4)       //]
								{
									z++;
									if (Token[z].tol == 7)       //-
									{
										z++;
										if (Token[z].tol == 8)       //->
										{
											z++;
											if (Token[z].tol == 3)      //[
											{
												z++;
												if (Token[z].tol == 4)       //]
												{
													z++;
													if (Token[z].tol == 6)      //)
													{

														z++;
														if (z == high)
														{
															SearchPro_A(irel, proname, A);
															int i = 0;
															while (i != pdos)
															{
																if (irel[i] == 1)
																{

																	printf("%s-[%s]->%s", A, proname, headnode[headpro[i].l].name);
																}
																i++;
															}
															break;
														}
													}
												}
											}
											else if (Token[z].tol == 0)      //str
											{
												char B[20];
												strc1(B, Token[z].str);
												z++;
												if (Token[z].tol == 6)      //)
												{
													z++;
													if (z == high)
													{
														printf("222222");
														SearchPro_AB(irel, proname, A, B);
														int i = 0;
														while (i != pdos)
														{
															if (irel[i] == 1)
															{
																printf("%s-[%s]->%s", A, proname, B);

															}
															i++;
														}
														break;
													}
												}

											}

										}

									}

								}

							}

						}

					}

				}
				else if (Token[z].tol == 3)      //[
				{
					z++;
					if (Token[z].tol == 4)         //]
					{
						z++;
						if (Token[z].tol == 7)        //-
						{
							z++;
							if (Token[z].tol == 3)       //[
							{
								z++;
								if (Token[z].tol == 0)       //str
								{
									char proname[20];
									strc1(proname, Token[z].str);
									z++;
									if (Token[z].tol == 4)        //]
									{
										z++;
										if (Token[z].tol == 7)       //-
										{
											z++;
											if (Token[z].tol == 8)       //>
											{
												z++;
												if (Token[z].tol == 3)      //[
												{
													z++;
													if (Token[z].tol == 4)      //]
													{
														z++;
														if (Token[z].tol == 6)      //)
														{
															z++;
															if (z == high)
															{
																printf("333333333");
																SearchPro(irel, proname);
																int i = 0;
																while (i != pdos)
																{
																	if (irel[i] == 1)
																	{
																		printf("%s-[%s]->%s", headnode[headpro[i].h].name, proname, headnode[headpro[i].l].name);

																	}
																	i++;
																}
																break;
															}

														}

													}

												}
												else if (Token[z].tol == 0)    //str
												{
													char B[20];
													strc1(B, Token[z].str);
													z++;
													if (Token[z].tol == 6)     //)
													{
														z++;
														if (z == high)
														{
															printf("44444444444");
															SearchPro_B(irel, proname, B);
															int i = 0;
															if (irel[i] == 1)
															{
																printf("%s-[%s]->%s", headnode[headpro[i].h].name, proname, B);

															}
															i++;
															break;
														}

													}

												}

											}

										}

									}

								}

							}

						}

					}

				}

			}
			
		}
		else if (strcmp(Token[z].str, c6) == 0)  //delete rel
		{
			char err[] = "[ input err! ]";
			int* irel = (int*)malloc(sizeof(int) * pdos);
			z++;
			if (Token[z].tol == 5)      //(
			{
				z++;
				if (Token[z].tol == 0)       //str
				{
					char A[20];
					strc1(A, Token[z].str);
					z++;
					if (Token[z].tol == 7)       //-
					{
						z++;
						if (Token[z].tol == 3)      //[
						{
							z++;
							if (Token[z].tol == 0)      //str
							{
								char proname[20];
								strc1(proname, Token[z].str);
								z++;
								if (Token[z].tol == 4)       //]
								{
									z++;
									if (Token[z].tol == 7)       //-
									{
										z++;
										if (Token[z].tol == 8)       //->
										{
											z++;
											if (Token[z].tol == 3)      //[
											{
												z++;
												if (Token[z].tol == 4)       //]
												{
													z++;
													if (Token[z].tol == 6)      //)
													{

														z++;
														if (z == high)
														{
															DeletPro_A(irel, proname, A);
														}
														break;
													}
												}
											}
											else if (Token[z].tol == 0)      //str
											{
												char B[20];
												strc1(B, Token[z].str);
												z++;
												if (Token[z].tol == 6)      //)
												{
													z++;
													if (z == high)
													{
														DeletPro_AB(irel, proname, A, B);

													}
													break;

												}

											}

										}

									}

								}

							}

						}

					}

				}
				else if (Token[z].tol == 3)      //[
				{
					z++;
					if (Token[z].tol == 4)         //]
					{
						z++;
						if (Token[z].tol == 7)        //-
						{
							z++;
							if (Token[z].tol == 3)       //[
							{
								z++;
								if (Token[z].tol == 0)       //str
								{
									char proname[20];
									strc1(proname, Token[z].str);
									z++;
									if (Token[z].tol == 4)        //]
									{
										z++;
										if (Token[z].tol == 7)       //-
										{
											z++;
											if (Token[z].tol == 8)       //>
											{
												z++;
												if (Token[z].tol == 3)      //[
												{
													z++;
													if (Token[z].tol == 4)      //]
													{
														z++;
														if (Token[z].tol == 6)      //)
														{
															z++;
															if (z == high)
															{
																DeletPro(irel, proname);
															}
															break;
														}

													}

												}
												else if (Token[z].tol == 0)    //str
												{
													char B[20];
													strc1(B, Token[z].str);
													z++;
													if (Token[z].tol == 6)     //)
													{
														z++;
														if (z == high)
														{
															DeletPro_B(irel, proname, B);
														}
														break;
													}

												}

											}

										}

									}

								}

							}

						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c5) == 0) //Deletenode
		{
			char err[] = "[ input err! ]";
			z++;
			if (Token[z].tol == 0)      //str
			{
				char nodename[20];
				strc1(nodename, Token[z].str);
				z++;
				if (Token[z].tol == 5)      //(
				{
					z++;
					if (strcmp(Token[z].str, c7) == 0)   //where
					{
						z++;
						if (strcmp(Token[z].str, c15) == 0)     //str
						{

							z++;
							if (Token[z].tol == 9)      //=
							{
								z++;
								if (Token[z].tol == 0)      //str
								{
									z++;
									char labname1[20];
									strc1(labname1, Token[z].str);
									if (Token[z].tol == 6)     //)
									{
										z++;
										if (z == high)
										{
											Deletnode(nodename, labname1);
											break;
										}

									}

								}

							}

						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c11) == 0)// updatenode
		{
			char err[] = "[ input err! ]";
			z++;
			if (Token[z].tol == 0)      //str
			{
				char nodename[20];
				int  inode;
				strc1(nodename, Token[z].str);
				z++;
				if (Token[z].tol == 5)      //(
				{
					z++;
					if (strcmp(Token[z].str, c7) == 0)   //where
					{
						z++;
						if (strcmp(Token[z].str, c14) == 0)     //labname
						{
							z++;
							if (Token[z].tol == 9)      //=
							{
								z++;
								if (Token[z].tol == 0)      //str
								{
									char labname1[20];
									strc1(labname1, Token[z].str);
									z++;
									if (Token[z].tol == 6)     //)
									{
										z++;
										if (Token[z].tol == 1)
										{
											z++;
											if (strcmp(Token[z].str, c13) == 0)
											{
												z++;
												if (Token[z].tol == 9)
												{
													z++;
													if (Token[z].tol == 0)
													{
														char newname[20];
														z++;
														if (z == high)
														{
															updatenodename(nodename, newname);
														}

													}

												}


											}

										}

									}

								}

							}

						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c12) == 0)  //update rel
		{
			char err[] = "[ input err! ]";
			//int* irel = (int*)malloc(sizeof(int) * pdos);
			z++;
			if (Token[z].tol == 5)      //(
			{
				z++;
				if (Token[z].tol == 0)       //str
				{
					char A[20];
					strc1(A, Token[z].str);
					z++;
					if (Token[z].tol == 7)       //-
					{
						z++;
						if (Token[z].tol == 3)      //[
						{
							z++;
							if (Token[z].tol == 0)      //str
							{
								char proname[20];
								strc1(proname, Token[z].str);
								z++;
								if (Token[z].tol == 4)       //]
								{
									z++;
									if (Token[z].tol == 7)       //-
									{
										z++;
										if (Token[z].tol == 8)       //->
										{
											z++;
											if (Token[z].tol == 3)      //[
											{
												z++;
												if (Token[z].tol == 4)       //]
												{
													z++;
													if (Token[z].tol == 6)      //)
													{

														z++;
														if (Token[z].tol == 1)
														{
															z++;
															if (Token[z].tol == 0)
															{
																char newname[20];
																strc1(newname, Token[z].str);
																if (z == high)
																{
																	updateProname_A(proname, A, newname);
																	printf("update success!");
																	break;
																}
															}
														}

													}
												}
											}
											else if (Token[z].tol == 0)      //str
											{
												char B[20];
												strc1(B, Token[z].str);
												z++;
												if (Token[z].tol == 6)      //)
												{
													z++;
													if (Token[z].tol == 1)
													{
														z++;
														if (Token[z].tol == 0)
														{
															char newname[20];
															strc1(newname, Token[z].str);
															if (z == high)
															{
																updateProname_AB(proname, A, B, newname);
																printf("update success!");
																break;
															}
														}
													}

												}

											}

										}

									}

								}

							}

						}

					}

				}
				else if (Token[z].tol == 3)      //[
				{
					z++;
					if (Token[z].tol == 4)         //]
					{
						z++;
						if (Token[z].tol == 7)        //-
						{
							z++;
							if (Token[z].tol == 3)       //[
							{
								z++;
								if (Token[z].tol == 0)       //str
								{
									char proname[20];
									strc1(proname, Token[z].str);
									z++;
									if (Token[z].tol == 4)        //]
									{
										z++;
										if (Token[z].tol == 7)       //-
										{
											z++;
											if (Token[z].tol == 8)       //>
											{
												z++;
												if (Token[z].tol == 3)      //[
												{
													z++;
													if (Token[z].tol == 4)      //]
													{
														z++;
														if (Token[z].tol == 6)      //)
														{
															z++;
															if (Token[z].tol == 1)
															{
																z++;
																if (Token[z].tol == 0)
																{
																	char newname[20];
																	strc1(newname, Token[z].str);
																	if (z == high)
																	{
																		updateProname(proname, newname);
																		printf("update success!");
																		break;
																	}
																}
															}

														}

													}

												}
												else if (Token[z].tol == 0)    //str
												{
													char B[20];
													strc1(B, Token[z].str);
													z++;
													if (Token[z].tol == 6)     //)
													{
														z++;
														if (Token[z].tol == 1)
														{
															z++;
															if (Token[z].tol == 0)
															{
																char newname[20];
																strc1(newname, Token[z].str);
																if (z == high)
																{
																	updateProname_B(proname, B, newname);
																	printf("update success!");
																	break;
																}
															}
														}

													}

												}

											}

										}

									}

								}

							}

						}

					}

				}

			}

		}
		else if (strcmp(Token[z].str, c22) == 0)
		{
			renew();
		}
		else if (strcmp(Token[z].str, c20) == 0)
		{
			ex = 0;
			printf("exit success!");
			break;
		}
		else if (strcmp(Token[z].str, c15) == 0)
		{
			//char  filename[] = "C:\\Users\\YuWenJian\\Desktop\\task+++";
			///Datainsert(filename);
			//char re[] = "insert finish!";
			//return re;
		}
		else
		{
			break;
		}
	}
}

int main()
{  
	while (ex) 
	{
		char input_cypher[100];
		printf("input cypher:");
		gets(input_cypher);
		cypher(input_cypher);
	}
	
	return 0;
}